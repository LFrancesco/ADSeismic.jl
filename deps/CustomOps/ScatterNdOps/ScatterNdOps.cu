#include "hip/hip_runtime.h"
#include <stdio.h>
__global__ void ScatterNdOps_forward_kernel(double *out, const long long*ii, const double *update, int n){
    int p =  blockIdx.x *blockDim.x + threadIdx.x;
    if (p<n){
        out[ii[p]-1] = update[p];
    }
}

__global__ void setzero_kernel(double *out, int n){
    int p =  blockIdx.x *blockDim.x + threadIdx.x;
    if (p<n){
        out[p] = 0.0;
    }
}

void Gpu_ScatterNdOps_forward(double *out, const long long *ii,
    const double *update, int n, int N){
    setzero_kernel<<< (N - 1)/64 + 1, 64 >>>(out, N);
    ScatterNdOps_forward_kernel<<< (n - 1)/64 + 1, 64 >>>(out, ii, update, n);
 }

 
 __global__ void ScatterNdOps_backward_kernel(double *grad_update, 
    const double *grad_out,
    const double *out, const long long *ii,
    const double *update, int n){
    int p =  blockIdx.x *blockDim.x + threadIdx.x;
    if (p<n) {
        grad_update[p] = grad_out[ii[p]-1];
    }
}

void Gpu_ScatterNdOps_backward(
    double *grad_update, 
    const double *grad_out,
    const double *out, const long long *ii,
    const double *update, int n){
    setzero_kernel<<< (n - 1)/64 + 1, 64 >>>(grad_update, n);
    ScatterNdOps_backward_kernel<<< (n - 1)/64 + 1, 64 >>>(grad_update, grad_out, out, ii, update, n);
 }

 void get_ScatterNdOps_num(long long *out, const long long *m){
    hipMemcpy(out, m, sizeof(long long), hipMemcpyDeviceToHost);
 }